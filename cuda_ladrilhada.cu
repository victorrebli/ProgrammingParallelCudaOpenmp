#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <omp.h>
#include <time.h>
#define Tile_size 64

int numARows = 1024;   // number of rows in the matrix A
int numAColumns = 1024;  // number of columns in the matrix A
int numBRows = 1024;   // number of rows in the matrix B
int numBColumns = 1024;  // number of columns in the matrix B
int numCRows = 1024;  // number of rows in the matrix C (you have to set this)
int numCColumns = 1024; // number of columns in the matrix C (you have to set this)


__global__ void matrixMultiplyShared(float * A, float * B, float * C,
                                    int numARows, int numAColumns,
                                    int numBRows, int numBColumns,
                                    int numCRows, int numCColumns)
{
    __shared__ float sA[Tile_size][Tile_size];   // Tile size to store elements in shared memory
    __shared__ float sB[Tile_size][Tile_size];

    int Row = blockDim.y*blockIdx.y + threadIdx.y; //To generate ids of threads.
    int Col = blockDim.x*blockIdx.x + threadIdx.x;
    float Cvalue = 0.0;
    sA[threadIdx.y][threadIdx.x] = 0.0;
    sB[threadIdx.y][threadIdx.x] = 0.0;

    for (int k = 0; k < (((numAColumns - 1)/ Tile_size) + 1); k++)
    {
        if ( (Row < numARows) && (threadIdx.x + (k*Tile_size)) < numAColumns)//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
        {
            sA[threadIdx.y][threadIdx.x] = A[(Row*numAColumns) + threadIdx.x + (k*Tile_size)];
        }
        else
        {
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }
        if ( Col < numBColumns && (threadIdx.y + k*Tile_size) < numBRows)//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
        {
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k*Tile_size)*numBColumns + Col];
        }
        else
        {
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }
        __syncthreads();

        for (int j = 0; j < Tile_size; ++j)//Multiplying Elements present in tile
        {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }
    }
    if (Row < numCRows && Col < numCColumns)//Saving Final result into Matrix C
    {
        C[Row*numCColumns + Col] = Cvalue;
    }
}
//*************************************************************
void Print_Mat(int Row,int Col,float * Mat)
{
 for(int i=0;i<Row*Col;i++)
   {
   printf("%f  ",*(Mat+i));

   if((i%Col)==0 )
    {
     printf("\n");
    }
   }
}//Function close

int main(int argc, char ** argv) {
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * hostComputedC;
    float * deviceA;
    float * deviceB;
    float * deviceC;
    int tamanho = numARows * numAColumns;
    float time_1;
    hipEvent_t start, stop;

    hostA = (float *) malloc(sizeof(float)*numARows*numAColumns);
    hostB = (float *) malloc(sizeof(float)*numBRows*numBColumns);

    for (int i = 0; i < numARows*numAColumns; i++)//Matrix Initialization
    {
        hostA[i]=((float)rand()/(float)(RAND_MAX)) * 3.0;
    }
    for (int i = 0; i < numBRows*numBColumns; i++)
    {
        hostB[i]=((float)rand()/(float)(RAND_MAX)) * 3.0;
    }

   
    numCRows = numARows;
    numCColumns = numBColumns;

    hostC = (float *) malloc(sizeof(float)*numCRows*numCColumns);
    hostComputedC = (float *) malloc(sizeof(float)*numCRows*numCColumns);

    
    hipMalloc((void **)&deviceA, sizeof(float)*numARows*numAColumns);
    hipMalloc((void **)&deviceB, sizeof(float)*numBRows*numBColumns);
    hipMalloc((void **)&deviceC, sizeof(float)*numCRows*numCColumns);

    
    hipMemcpy(deviceA, hostA, sizeof(float)*numARows*numAColumns, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, sizeof(float)*numBRows*numBColumns, hipMemcpyHostToDevice);

   

    dim3 dimGrid((numCColumns/Tile_size) + 1, (numCRows/Tile_size) + 1, 1);//Number of Blocks required
    dim3 dimBlock(Tile_size, Tile_size, 1);//Number of threads in each block

    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, NULL);
    int iter = 1000;
    for (int j =0; j<iter; j++)
    {
    //printf("chamada: %d \n", j);
    matrixMultiplyShared<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
    }

    hipEventRecord( stop, NULL);
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time_1, start, stop );

    hipError_t err1 = hipPeekAtLastError();

    hipDeviceSynchronize();

    
    hipMemcpy(hostC, deviceC, sizeof(float)*numCRows*numCColumns, hipMemcpyDeviceToHost);

    printf("Effective Bandwidth (GB/s): %f \n", 2*tamanho/time_1/1e6);
    float msecPerMatrixMul = time_1;
    double flopsPerMatrixMul = 2.0 * (double)tamanho;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul);

    // Free the GPU memory
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
    //Free the Pointer Memory
    free(hostA);
    free(hostB);
    free(hostC);
    free(hostComputedC);

    return 0;
}
